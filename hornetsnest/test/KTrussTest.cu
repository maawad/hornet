#include "Static/KTruss/KTruss.cuh"

#include <StandardAPI.hpp>
#include <Device/Util/Timer.cuh>
#include <Graph/GraphStd.hpp>

using namespace std;
using namespace timer;
using namespace hornets_nest;


int main(int argc, char **argv) {

    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;

    int device=0;

    hipSetDevice(device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
 
    graph::GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    graph.read(argv[1], SORT | PRINT_INFO);


    HornetInit hornet_init(graph.nV(), graph.nE(),
                                 graph.csr_out_offsets(),
                                 graph.csr_out_edges());

    std::cout << "Initializing GPU graph" << std::endl;
    HornetGraph hornet_graph(hornet_init);
    std::cout << "Checking sortd adj" << std::endl;

    hornet_graph.check_sorted_adjs();
    // std::cout << "Is sorted " <<  << std::endl;

    KTruss ktruss(hornet_graph);

    ktruss.init();
    ktruss.reset();
    ktruss.run();
}
