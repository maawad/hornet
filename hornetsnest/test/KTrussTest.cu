#include "Static/KTruss/KTruss.cuh"
#include <Device/Util/Timer.cuh>
#include <Graph/GraphStd.hpp>

using namespace timer;
using namespace hornets_nest;

int main(int argc, char **argv) {
    // hipSetDevice(1);
    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;

    graph::GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    graph.read(argv[1], SORT | PRINT_INFO);

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges(), true);

    HornetInit hcopy_init(graph.nV(), 0, graph.csr_out_offsets(),
                           graph.csr_out_edges(), true);

    HornetGraph hornet_graph(hornet_init);
    KTruss ktruss(hornet_graph);
    ktruss.run();
}
